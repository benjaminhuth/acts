// This file is part of the ACTS project.
//
// Copyright (C) 2016 CERN for the benefit of the ACTS project
//
// This Source Code Form is subject to the terms of the Mozilla Public
// License, v. 2.0. If a copy of the MPL was not distributed with this
// file, You can obtain one at https://mozilla.org/MPL/2.0/.

#include "Acts/Plugins/ExaTrkX/CudaTrackBuilding.hpp"
#include "Acts/Plugins/ExaTrkX/detail/ConnectedComponents.cuh"
#include "Acts/Plugins/ExaTrkX/detail/CudaUtils.cuh"
#include "Acts/Plugins/ExaTrkX/detail/CudaUtils.hpp"
#include "Acts/Plugins/ExaTrkX/detail/JunctionRemoval.hpp"
#include "Acts/Utilities/Zip.hpp"

namespace Acts {

std::pair<Acts::Tensor<int>, std::size_t> CudaTrackBuilding::operator()(
    PipelineTensors tensors, const ExecutionContext& execContext) {
  ACTS_VERBOSE("Start CUDA track building");
  if (!(tensors.edgeIndex.device().isCuda() &&
        tensors.edgeScores.value().device().isCuda())) {
    throw std::runtime_error(
        "CudaTrackBuilding expects tensors to be on CUDA!");
  }

  const auto numSpacepoints = tensors.nodeFeatures.shape().at(0);
  auto numEdges = tensors.edgeIndex.shape().at(1);

  if (numEdges == 0) {
    ACTS_DEBUG("No edges remained after edge classification");
    return {Acts::Tensor<int>::Create({0, 0}, execContext), 0};
  }

  auto stream = execContext.stream.value();

  auto cudaSrcPtr = tensors.edgeIndex.data();
  auto cudaTgtPtr = tensors.edgeIndex.data() + numEdges;

  auto ms = [](auto t0, auto t1) {
    return std::chrono::duration_cast<std::chrono::milliseconds>(t1 - t0)
        .count();
  };

  if (m_cfg.doJunctionRemoval) {
    assert(tensors.edgeScores->shape().at(0) ==
           tensors.edgeIndex.shape().at(1));
    auto cudaScorePtr = tensors.edgeScores->data();

    ACTS_DEBUG("Do junction removal...");
    auto t0 = std::chrono::high_resolution_clock::now();
    auto [cudaSrcPtrJr, numEdgesOut] = detail::junctionRemovalCuda(
        numEdges, numSpacepoints, cudaScorePtr, cudaSrcPtr, cudaTgtPtr, stream);
    auto t1 = std::chrono::high_resolution_clock::now();
    cudaSrcPtr = cudaSrcPtrJr;
    cudaTgtPtr = cudaSrcPtrJr + numEdgesOut;

    if (numEdgesOut == 0) {
      ACTS_WARNING(
          "No edges remained after junction removal, this should not happen!");
      ACTS_CUDA_CHECK(hipFreeAsync(cudaSrcPtrJr, stream));
      ACTS_CUDA_CHECK(hipStreamSynchronize(stream));
      return {Acts::Tensor<int>::Create({0, 0}, execContext), 0};
    }

    ACTS_DEBUG("Removed " << numEdges - numEdgesOut
                          << " edges in junction removal");
    ACTS_DEBUG("Junction removal took " << ms(t0, t1) << " ms");
    numEdges = numEdgesOut;
  }

  auto cudaLabels = Acts::Tensor<int>::Create({numSpacepoints, 1}, execContext);

  auto t0 = std::chrono::high_resolution_clock::now();
  std::size_t numberLabels = detail::connectedComponentsCuda(
      numEdges, cudaSrcPtr, cudaTgtPtr, numSpacepoints, cudaLabels.data(),
      stream, m_cfg.useOneBlockImplementation);
  auto t1 = std::chrono::high_resolution_clock::now();
  ACTS_DEBUG("Connected components took " << ms(t0, t1) << " ms");

  // TODO not sure why there is an issue that is not detected in the unit tests
  numberLabels += 1;

  // Free Memory
  if (m_cfg.doJunctionRemoval) {
    ACTS_CUDA_CHECK(hipFreeAsync(cudaSrcPtr, stream));
  }

  return {std::move(cudaLabels), numberLabels};
}

}  // namespace Acts
