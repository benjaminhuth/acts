// This file is part of the ACTS project.
//
// Copyright (C) 2016 CERN for the benefit of the ACTS project
//
// This Source Code Form is subject to the terms of the Mozilla Public
// License, v. 2.0. If a copy of the MPL was not distributed with this
// file, You can obtain one at https://mozilla.org/MPL/2.0/.

#include "Acts/Plugins/ExaTrkX/CudaTrackBuilding.hpp"
#include "Acts/Plugins/ExaTrkX/detail/ConnectedComponents.cuh"
#include "Acts/Plugins/ExaTrkX/detail/CudaUtils.cuh"
#include "Acts/Plugins/ExaTrkX/detail/CudaUtils.hpp"
#include "Acts/Plugins/ExaTrkX/detail/JunctionRemoval.hpp"
#include "Acts/Utilities/Zip.hpp"

namespace Acts {

std::vector<std::vector<int>> CudaTrackBuilding::operator()(
    PipelineTensors tensors, std::vector<int>& spacepointIDs,
    const ExecutionContext& execContext) {
  ACTS_VERBOSE("Start CUDA track building");
  if (!(tensors.edgeIndex.device().isCuda() &&
        tensors.edgeScores.value().device().isCuda())) {
    throw std::runtime_error(
        "CudaTrackBuilding expects tensors to be on CUDA!");
  }

  const auto numSpacepoints = spacepointIDs.size();
  auto numEdges = static_cast<std::size_t>(tensors.edgeIndex.shape().at(1));

  if (numEdges == 0) {
    ACTS_DEBUG("No edges remained after edge classification");
    return {};
  }

  auto stream = execContext.stream.value();

  auto cudaSrcPtr = tensors.edgeIndex.data();
  auto cudaTgtPtr = tensors.edgeIndex.data() + numEdges;

  auto t0 = ACTS_TIME_STREAM_SYNC(Acts::Logging::DEBUG, execContext.stream);

  if (m_cfg.doJunctionRemoval) {
    assert(tensors.edgeScores->shape().at(0) ==
           tensors.edgeIndex.shape().at(1));
    auto cudaScorePtr = tensors.edgeScores->data();

    ACTS_DEBUG("Do junction removal...");
    auto [cudaSrcPtrJr, numEdgesOut] = detail::junctionRemovalCuda(
        numEdges, numSpacepoints, cudaScorePtr, cudaSrcPtr, cudaTgtPtr, stream);
    cudaSrcPtr = cudaSrcPtrJr;
    cudaTgtPtr = cudaSrcPtrJr + numEdgesOut;

    if (numEdgesOut == 0) {
      ACTS_WARNING(
          "No edges remained after junction removal, this should not happen!");
      ACTS_CUDA_CHECK(hipFreeAsync(cudaSrcPtrJr, stream));
      ACTS_CUDA_CHECK(hipStreamSynchronize(stream));
      return {};
    }

    ACTS_DEBUG("Removed " << numEdges - numEdgesOut
                          << " edges in junction removal");
    numEdges = numEdgesOut;
  }

  auto t1 = ACTS_TIME_STREAM_SYNC(Acts::Logging::DEBUG, execContext.stream);

  int* cudaLabels{};
  ACTS_CUDA_CHECK(
      hipMallocAsync(&cudaLabels, numSpacepoints * sizeof(int), stream));

  std::size_t numberLabels = detail::connectedComponentsCuda(
      numEdges, cudaSrcPtr, cudaTgtPtr, numSpacepoints, cudaLabels, stream,
      m_cfg.useOneBlockImplementation);

  auto t2 = ACTS_TIME_STREAM_SYNC(Acts::Logging::DEBUG, execContext.stream);

  // TODO not sure why there is an issue that is not detected in the unit tests
  numberLabels += 1;

  std::vector<int> trackLabels(numSpacepoints);
  ACTS_CUDA_CHECK(hipMemcpyAsync(trackLabels.data(), cudaLabels,
                                  numSpacepoints * sizeof(int),
                                  hipMemcpyDeviceToHost, stream));

  // Free Memory
  ACTS_CUDA_CHECK(hipFreeAsync(cudaLabels, stream));
  if (m_cfg.doJunctionRemoval) {
    ACTS_CUDA_CHECK(hipFreeAsync(cudaSrcPtr, stream));
  }

  ACTS_CUDA_CHECK(hipStreamSynchronize(stream));
  ACTS_CUDA_CHECK(hipGetLastError());

  auto t3 = ACTS_TIME_STREAM_SYNC(Acts::Logging::DEBUG, execContext.stream);

  ACTS_VERBOSE("Found " << numberLabels << " track candidates");

  std::vector<std::vector<int>> trackCandidates(numberLabels);

  for (const auto [label, id] : Acts::zip(trackLabels, spacepointIDs)) {
    trackCandidates[label].reserve(32);
    trackCandidates[label].push_back(id);
  }

  auto t4 = ACTS_TIME_STREAM_SYNC(Acts::Logging::DEBUG, execContext.stream);

  auto ms = [](auto t0, auto t1) {
    return std::chrono::duration<double, std::milli>(t1 - t0).count();
  };

  ACTS_DEBUG("Junction removal:     " << ms(t0, t1) << " ms");
  ACTS_DEBUG("Connected components: " << ms(t1, t2) << " ms");
  ACTS_DEBUG("Copy to host:         " << ms(t2, t3) << " ms");
  ACTS_DEBUG("Build vector<vector>: " << ms(t3, t4) << " ms");

  return trackCandidates;
}

}  // namespace Acts
